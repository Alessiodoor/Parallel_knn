#include "hip/hip_runtime.h"
#include "knnCuda.h" 
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <math.h>

/*
Funzione kernel per calcolare la distanza euclidea tra gli attributi di un sample di train e uno di test
Parametri:
N: numero sample di train
M: numero sample di test
A: numero di attributi
dev_train: sample di train sul device
dev_test: sample di testsul device
dev_distances: matrice delle distanze sul device
*/
__global__ void euclideanDistance_kernel(int N, int M, int A, const float* __restrict__ dev_train, const float* __restrict__ dev_test, float* __restrict__ dev_distances){//, int* dev_labels){
	
	// indice di inizio della riga
	int idx = threadIdx.x+blockDim.x*blockIdx.x;
  	int idy = threadIdx.y+blockDim.y*blockIdx.y;
	
	// controllo che gli indici del thread siano corretti
	if(idx < N && idy < M){
		float sum = 0.f;
		// rendo parallela l'esecuzione del for, tra i threads del blocco
	    #pragma unroll
	    for (int d = 0; d < A; ++d) {
	    	float x = dev_train[idx*A +d];  
	    	float y = dev_test[idy*A +d];
	        float diff = x - y;
	        sum += diff * diff;
	    }
		dev_distances[(idy * N) + idx] = sqrtf(sum);
	}
}

/*
Funzione kernel per ordinare ordinare i sample di train per il test i-esimo in base alla distanza tra train e test
Questa funzione ordina i vicini di un solo sample di test.
Parametri:
distances: array delle distanze tra il sample di test i-esimo e tutti i sample di train
index: array degli indici di tutti i sample di train per il sample id tst i-esimo
N: numero di sample di train
M: numero sample di test
K: numero di vicini
dev_distances: matrice delle distanze sul device
dev_labels: label del sample di test 
*/
__global__ void sort_kernel(int N, int M, int K, float* __restrict__ dev_distances, int* __restrict__ dev_labels){
	
	// indice di inizio della riga
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	// controllo che l'indici sia corretto
	if(index < M){
		dev_labels[index * K] = 0;
		// rendo parallela l'esecuzione del for, tra i threads del blocco
		#pragma unroll
		for(int i=1; i< N; i++){
			float distanzaCorrente = dev_distances[index*N+i];
        	int indiceCorrente = i;
        	if( i >= K && distanzaCorrente >= dev_distances[index*N+ K-1]){
            	continue;
        	}
			
			int j = i;
        	if (j > K-1)
            	j = K-1;
        
        	while(j > 0 && dev_distances[index*N+ j-1] > distanzaCorrente){
            	dev_distances[index*N +j] = dev_distances[index*N+j-1];
            	dev_labels[index*K+j] = dev_labels[index*K+j-1];
            	--j;
        	}

        	dev_distances[index*N+j] = distanzaCorrente;
        	dev_labels[index*K+j] = indiceCorrente;	
		}
	}
}