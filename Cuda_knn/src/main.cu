#include <stdint.h> 
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

//#include "input.h"
#include "knnCuda.h"
//#include "check.h"
#include "hipError_t.h"
#include "datasetFunctions.h"

// attributi
#define A 30
// labels
#define LABELS 10
// numtreads
#define NT 32
//#define BLOCK_SIZE 32

int main(int argc, char *argv[])
{

	// argomenti:
   	// train file name
   	// test file name
   	// N: numero di sample di train
   	// M: numero di sample di test
   	// k: numero di vicini
   	// BLOCK_SIZE: numro di blocchi per grid
   	if(argc != 7){
      	printf(
        	 "Errore non sono stati specificati correttamente i parametri:\n"
         	"1 - Train fileName\n"
         	"2 - Test tileName\n"
         	"3 - Numero sample di train\n"
         	"4 - Numero sample di test\n"
         	"5 - K: numero di vicini\n"
         	"6 - BLOCK_SIZE: numero di blocchi per grid");
      	exit(EXIT_FAILURE);
   	}

   	const char * trainFile = argv[1];
   	const char * testFile = argv[2];

   	int N = atoi(argv[3]);
   	int M = atoi(argv[4]);
   	int K = atoi(argv[5]);
   	int BLOCK_SIZE = atoi(argv[6]);

   	if (K > N){
      	printf("Errore il numero di vicini non può essere superiore al numero di sample!\n");
      	exit(EXIT_FAILURE);
   	}

   	if (K % 2 == 0){
      	printf("Inserire un numero di vicini dispari!\n");
      	exit(EXIT_FAILURE);
   	}

	// device
	int deviceIndex = 0;

	// ottengo il numero di schede presenti
	int count;
	HANDLE_ERROR( hipGetDeviceCount( &count ) );
    
    // controllo l'esistenza della scheda disponbile
    if(deviceIndex < count)
    {
        HANDLE_ERROR(hipSetDevice(deviceIndex));
    }
    else
    {
        printf("Device non disponbile!\n");
        exit(EXIT_FAILURE);        
    }

    // proprietà della scheda video
    hipDeviceProp_t prop;
    HANDLE_ERROR(hipGetDeviceProperties(&prop, deviceIndex));

    // misuro il tempo d'esecuzione
	hipEvent_t start, stop, stopRead, stopSendData, primoStep, secondoStep;
	
	HANDLE_ERROR( hipEventCreate( &start ) );
	HANDLE_ERROR( hipEventCreate( &stop ) );
	HANDLE_ERROR( hipEventCreate( &stopRead ) );
	HANDLE_ERROR( hipEventCreate( &stopSendData ) );
	HANDLE_ERROR( hipEventCreate( &primoStep ) );
	HANDLE_ERROR( hipEventCreate( &secondoStep ) );
	
	HANDLE_ERROR( hipEventRecord( start, 0 ) );

	// alloco le matrici del dataset e degli altri vettori utili
	float * trainingData= (float *) malloc(N * A * sizeof(float));
	float * testingData= (float *) malloc(M * A * sizeof(float));

	int * classesTraining = (int*) malloc(N *sizeof(int));
	int * classesTesting = (int*)  malloc(M *sizeof(int));

	float * dist = (float *) malloc(M * N * sizeof(float));
	
	// controllo che le variabili siano state allocate correttamente
	if(trainingData == NULL || testingData == NULL || classesTesting == NULL || classesTraining == NULL){
		printf("Not enough memory!\n");
		exit(EXIT_FAILURE);
	}

	// leggo il dataset dal file
	readFile(trainFile, N, A, trainingData, classesTraining);
	readFile(testFile, M, A, testingData, classesTesting);

	HANDLE_ERROR( hipEventRecord( stopRead, 0 ) );
	HANDLE_ERROR( hipEventSynchronize( stopRead ) );
	
	// puntattori ai dati sul device
	float* dev_train; 
	float* dev_test;
	float* dev_dist;
	int* dev_label;
	
	// alloco la memoria per il dataset sulla gpu in memoria globale
	HANDLE_ERROR( hipMalloc((void**)&dev_train, N * A * sizeof(float)));
	
	HANDLE_ERROR( hipMalloc((void**)&dev_test, M * A * sizeof(float)));

	// allocco la matrice delle distanze e delle label
	HANDLE_ERROR( hipMalloc((void**)&dev_dist, N * M * sizeof(float)));

	// copio il cotenuto del dataset sulle variabili presenti sul device
	HANDLE_ERROR( hipMemcpy(dev_train, trainingData, N * A * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR( hipMemcpy(dev_test, testingData, M * A * sizeof(float), hipMemcpyHostToDevice));	

	HANDLE_ERROR( hipEventRecord(stopSendData, 0));
	HANDLE_ERROR( hipEventSynchronize(stopSendData));
	
	// registre il tempo di lettura
	float elapsedTimeRead;
	HANDLE_ERROR( hipEventElapsedTime(&elapsedTimeRead, start, stopSendData ));
	
	// creo i blocchi da BLOCK_SIZE * BLOCK_SIZE thread
	dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1); 

	// calcolo il numero di righe e colonne in base a BLOCK_SIZE
	int dim_row = (M + 1 % BLOCK_SIZE == 0) ? M / BLOCK_SIZE : M / BLOCK_SIZE + 1;
	int dim_col = (N + 1 % BLOCK_SIZE == 0) ? N / BLOCK_SIZE : N / BLOCK_SIZE + 1;
	
	// creo la griglia di threads
	dim3 grid(dim_col, dim_row, 1); 

	// calcola distanza euclidea tra i punti train e test attraverso la funzione kernel
	euclideanDistance_kernel<<<grid, block>>>(N, M, A, dev_train, dev_test, dev_dist);

	// alloco le variabili che mi serviranno per calcolare al matrice di confusione
	int * label = (int*) malloc(M * K *sizeof(int));
	int* countsLabel = (int*) malloc(sizeof(int)* LABELS);
	int* confusionMatrix = (int*) malloc(sizeof(int)* LABELS * LABELS);

	if(confusionMatrix ==NULL || countsLabel == NULL || label == NULL){
		printf("Not enough memory!\n");
		exit(EXIT_FAILURE);
	}

	// inizializza a zero la matrice di confusione
	for(int i = 0; i < LABELS * LABELS; i++){
		confusionMatrix[i] = 0;
	}

	// barriera per assicurarsi che tutte le distanze siano state calcolate
	hipDeviceSynchronize();
	HANDLE_ERROR( hipEventRecord(  primoStep, 0 ) );
	HANDLE_ERROR( hipEventSynchronize(  primoStep ) );
	
	// calcolo il tempo d'esecuzione
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTimeRead, start, primoStep ) );
	
	// elimino il dataset su device, perchè non verrà più usato
	HANDLE_ERROR( hipFree(dev_train) );
    HANDLE_ERROR( hipFree(dev_test) );

    // inizializzo l'array delle label sul device
    HANDLE_ERROR( hipMalloc( (void**)&dev_label, M * K * sizeof(int) ) );

    // creo 
	dim3 blockSort(BLOCK_SIZE, 1, 1);
	dim3 gridSort(dim_row, 1, 1);

	sort_kernel<<<gridSort, blockSort>>>(N, M, K, dev_dist, dev_label);

	// barriera per assicurarsi che tutti i threads abbiamo concluso l'operazione
	hipDeviceSynchronize();

	// copio l'array delle label dal device alla memoria principale
	HANDLE_ERROR(hipMemcpy(label , dev_label, M * K * sizeof(int), hipMemcpyDeviceToHost ) );
	
	HANDLE_ERROR( hipEventRecord(  secondoStep, 0 ) );
	HANDLE_ERROR( hipEventSynchronize(  secondoStep ) );
	
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTimeRead, start, secondoStep ) );
	
	// numero di errori di classificazione commessi dall'algoritmo KNN
	int error = 0;
	
	// calcolo la matrice di confusione, lasciato alla CPU
	// ciclo sui samples di test
	for (int i = 0; i < M; i++){
		// inizializzo l'array utilizzato per contare l'occorrenza delle labels
		for(int l = 0; l < LABELS; l++){
			countsLabel[l] = 0;
		}
		int bestLabel = 0;
		// ciclo sui k sample di train vicini al sample di test i-esimo
		for(int j = 0; j < K; j++){	
			// indice e classe del sample di train j-esimo
			int indice = label[i*K + j];
			int classe = classesTraining[indice]; 
			// incremento il contatore di questa classe
			countsLabel[classe] = countsLabel[classe] + 1;
			// aggiorno la classe migliore se il numero di occorrenze è maggiore
			if(countsLabel[classe] > countsLabel[bestLabel])
				bestLabel = classe;
			}

		// controllo che la label calcolata corrisponda a quella vera
		int realLabel = classesTesting[i];
		if (realLabel != bestLabel){
			error = error + 1;
		}
			
		// aggiorno la matrice di confusione
		confusionMatrix[realLabel * LABELS + bestLabel] = confusionMatrix[realLabel * LABELS + bestLabel] + 1;	
	}

	// libero memoria utilizzata
	free(trainingData); trainingData = NULL;
	free(testingData); testingData = NULL;
	free(dist); dist=NULL;
	
	free(classesTraining); classesTraining = NULL;
	free(classesTesting); classesTesting = NULL;
	
	free(confusionMatrix); confusionMatrix=NULL;
	
	free(label); label=NULL;
	free(countsLabel); countsLabel= NULL;

	// libero memoria sul device
	HANDLE_ERROR( hipFree(dev_label ) );
    HANDLE_ERROR( hipFree(dev_dist ) );
    	
	// calcol il tempo totale d'esecuzione
	HANDLE_ERROR( hipEventRecord( stop, 0 ) );
	HANDLE_ERROR( hipEventSynchronize( stop ) );
	
	float elapsedTime;
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
	
	HANDLE_ERROR( hipEventDestroy( start ) );
	HANDLE_ERROR( hipEventDestroy( stop ) );

	// salvo risultati su file
	saveResultsOnFile(K, N, M, A, elapsedTime/1000,BLOCK_SIZE);

	return 0;
}