#include <stdint.h> 
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

//#include "input.h"
#include "knnCuda.h"
//#include "check.h"
#include "hipError_t.h"
#include "datasetFunctions.h"

// attributi
#define A 30
// labels
#define LABELS 10
// numtreads
#define NT 32
//#define BLOCK_SIZE 32

int main(int argc, char *argv[])
{

	// argomenti:
   	// train file name
   	// test file name
   	// N: numero di sample di train
   	// M: numero di sample di test
   	// k: numero di vicini
   	if(argc != 7){
      	printf(
        	 "Errore non sono stati specificati correttamente i parametri:\n"
         	"1 - Train fileName\n"
         	"2 - Test tileName\n"
         	"3 - Numero sample di train\n"
         	"4 - Numero sample di test\n"
         	"5 - K: numero di vicini\n"
         	"6 - BLOCK_SIZE: numero di blocchi per grid");
      	exit(EXIT_FAILURE);
   	}

   	const char * trainFile = argv[1];
   	const char * testFile = argv[2];

   	int N = atoi(argv[3]);
   	int M = atoi(argv[4]);
   	int K = atoi(argv[5]);
   	int BLOCK_SIZE = atoi(argv[6]);

   	if (K > N){
      	printf("Errore il numero di vicini non può essere superiore al numero di sample!\n");
      	exit(EXIT_FAILURE);
   	}

   	if (K % 2 == 0){
      	printf("Inserire un numero di vicini dispari!\n");
      	exit(EXIT_FAILURE);
   	}

	//device
	int deviceIndex = 0;

	//numero di schede presenti
	int count;
	HANDLE_ERROR( hipGetDeviceCount( &count ) );
    
    //check esistenza scheda disponbile
    if(deviceIndex < count)
    {
        HANDLE_ERROR(hipSetDevice(deviceIndex));
    }
    else
    {
        printf("Device non disponbile!\n");
        exit(EXIT_FAILURE);        
    }

    // proprietà della scheda video
    hipDeviceProp_t prop;
    HANDLE_ERROR(hipGetDeviceProperties(&prop, deviceIndex));

    // misurare il tempo di esecuzione
	hipEvent_t start, stop, stopRead, stopSendData, primoStep, secondoStep;
	HANDLE_ERROR( hipEventCreate( &start ) );
	HANDLE_ERROR( hipEventCreate( &stop ) );
	HANDLE_ERROR( hipEventCreate( &stopRead ) );
	HANDLE_ERROR( hipEventCreate( &stopSendData ) );
	HANDLE_ERROR( hipEventCreate( &primoStep ) );
	HANDLE_ERROR( hipEventCreate( &secondoStep ) );
	
	HANDLE_ERROR( hipEventRecord( start, 0 ) );

	// alloco dataset
	float * trainingData= (float *) malloc(N * A * sizeof(float));
	float * testingData= (float *) malloc(M * A * sizeof(float));

	int * classesTraining = (int*) malloc(N *sizeof(int));
	int * classesTesting = (int*)  malloc(M *sizeof(int));

	float * dist = (float *) malloc(M * N * sizeof(float));
	
	if(trainingData == NULL || testingData == NULL || classesTesting == NULL || classesTraining == NULL){
		printf("Not enough memory!\n");
		exit(EXIT_FAILURE);
	}

	//read dataset from file
	readFile(trainFile, N, A, trainingData, classesTraining);
	readFile(testFile, M, A, testingData, classesTesting);

	// get stop time, and display the timing results
	HANDLE_ERROR( hipEventRecord( stopRead, 0 ) );
	HANDLE_ERROR( hipEventSynchronize( stopRead ) );
	
	// puntattori ai dati sul device
	float* dev_train; 
	
	float* dev_test;

	float* dev_dist;

	int* dev_label;
	
	
	// alloco memoria per il dataset sulla gpu in memoria globale
	HANDLE_ERROR( hipMalloc((void**)&dev_train, N * A * sizeof(float)));
	
	HANDLE_ERROR( hipMalloc((void**)&dev_test, M * A * sizeof(float)));

	//allocco matrice distanze e relative label
	HANDLE_ERROR( hipMalloc((void**)&dev_dist, N * M * sizeof(float)));

	// copia elementi del dataset
	HANDLE_ERROR( hipMemcpy(dev_train, trainingData, N * A * sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR( hipMemcpy(dev_test, testingData, M * A * sizeof(float), hipMemcpyHostToDevice));	
	//HANDLE_ERROR( hipMemcpy( dev_dist, dist, N * P * sizeof(float), hipMemcpyHostToDevice ) );

	HANDLE_ERROR( hipEventRecord(stopSendData, 0));
	HANDLE_ERROR( hipEventSynchronize(stopSendData));
	
	float elapsedTimeRead;
	HANDLE_ERROR( hipEventElapsedTime(&elapsedTimeRead, start, stopSendData ));
	
	// creo blocchi da BLOCK_SIZE * BLOCK_SIZE thread
	dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1); 

	//Numero di blocchi
	int dim_row = (M + 1 % BLOCK_SIZE == 0) ? M / BLOCK_SIZE : M / BLOCK_SIZE + 1;
	int dim_col = (N + 1 % BLOCK_SIZE == 0) ? N / BLOCK_SIZE : N / BLOCK_SIZE + 1;
	
	dim3 grid(dim_col, dim_row, 1); // a grid of CUDA thread blocks

	// calcola distanza euclidea tra punti train e test
	euclideanDistance_kernel<<<grid, block>>>(N, M, A, dev_train, dev_test, dev_dist);//, dev_label);

	int * label = (int*) malloc(M * K *sizeof(int));
	int* countsLabel = (int*) malloc(sizeof(int)* LABELS);
	int* confusionMatrix = (int*) malloc(sizeof(int)* LABELS * LABELS);

	if(confusionMatrix ==NULL || countsLabel == NULL || label == NULL){
		printf("Not enough memory!\n");
		exit(EXIT_FAILURE);
	}

	// inizializza a zero la matrice di confusione
	for(int i = 0; i < LABELS * LABELS; i++){
		confusionMatrix[i] = 0;
	}

	// barriera per assicurarsi che tutte le distanze siano state calcolate
	hipDeviceSynchronize();
	HANDLE_ERROR( hipEventRecord(  primoStep, 0 ) );
	HANDLE_ERROR( hipEventSynchronize(  primoStep ) );
	
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTimeRead, start, primoStep ) );
	
	//dealloco dataset su device non più utile
	HANDLE_ERROR( hipFree(dev_train) );
    HANDLE_ERROR( hipFree(dev_test) );

    // sort
    HANDLE_ERROR( hipMalloc( (void**)&dev_label, M * K * sizeof(int) ) );

	dim3 blockSort(BLOCK_SIZE, 1, 1);
	dim3 gridSort(dim_row, 1, 1);

	sort_kernel<<<gridSort, blockSort>>>(N, M, K, dev_dist, dev_label);

	hipDeviceSynchronize();

	//recupero risultati dalla GPU
	HANDLE_ERROR(hipMemcpy(label , dev_label, M * K * sizeof(int), hipMemcpyDeviceToHost ) );
	
	HANDLE_ERROR( hipEventRecord(  secondoStep, 0 ) );
	HANDLE_ERROR( hipEventSynchronize(  secondoStep ) );
	
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTimeRead, start, secondoStep ) );
	
	// numero di errori di classificazione commessi dall'algoritmo KNN
	int error = 0;
	
	//il calcolo della matrice di confusione finale viene lasciato alla cpu
	for (int i = 0; i < M; i++){
		for(int l = 0; l < LABELS; l++){
			countsLabel[l] = 0;
		}
		int bestLabel = 0;
		for(int j = 0; j < K; j++){	
			int indice = label[i*K + j];
			int classe = classesTraining[indice]; 
			countsLabel[classe] = countsLabel[classe] + 1;
			if(countsLabel[classe] > countsLabel[bestLabel])
				bestLabel = classe;
			}

		int realLabel = classesTesting[i];
		if (realLabel != bestLabel){
			error = error + 1;
		}
			
		//update confusion matrix
		confusionMatrix[realLabel * LABELS + bestLabel] = confusionMatrix[realLabel * LABELS + bestLabel] + 1;	
	}

	// libero memoria 
	free(trainingData); trainingData = NULL;
	free(testingData); testingData = NULL;
	free(dist); dist=NULL;
	
	free(classesTraining); classesTraining = NULL;
	free(classesTesting); classesTesting = NULL;
	
	free(confusionMatrix); confusionMatrix=NULL;
	
	free(label); label=NULL;
	free(countsLabel); countsLabel= NULL;

	// libero memoria GPU
	HANDLE_ERROR( hipFree(dev_label ) );
    HANDLE_ERROR( hipFree(dev_dist ) );
    	
	// conteggio tempo totale di esecuzione
	HANDLE_ERROR( hipEventRecord( stop, 0 ) );
	HANDLE_ERROR( hipEventSynchronize( stop ) );
	
	float elapsedTime;
	HANDLE_ERROR( hipEventElapsedTime( &elapsedTime, start, stop ) );
	
	//printf( "Total time: %f \n", elapsedTime/1000 );
	HANDLE_ERROR( hipEventDestroy( start ) );
	HANDLE_ERROR( hipEventDestroy( stop ) );
	//HANDLE_ERROR( hipEventDestroy( stopRead ) );

	// salvo risultati su json
	saveResultsOnFile(K, N, M, A, elapsedTime/1000,BLOCK_SIZE);
    //writeResultJson(K, N, M, A, elapsedTime/1000, "resultCuda.json");

	return 0;
}